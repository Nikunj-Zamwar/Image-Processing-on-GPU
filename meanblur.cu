#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Function to load a PGM file
void loadPGMub(const char *filename, unsigned char **pixels, unsigned int *width, unsigned int *height) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        fprintf(stderr, "Error: Unable to open file %s\n", filename);
        exit(1);
    }

    // Read PGM header
    char magic[3];
    fscanf(file, "%s", magic);
    if (strcmp(magic, "P5") != 0) {
        fprintf(stderr, "Error: Invalid PGM file format\n");
        fclose(file);
        exit(1);
    }

    fscanf(file, "%u", width);
    fscanf(file, "%u", height);
    unsigned int max_val;
    fscanf(file, "%u%*c", &max_val);

    // Allocate memory for image pixels
    *pixels = (unsigned char *)malloc((*width) * (*height) * sizeof(unsigned char));
    if (!*pixels) {
        fprintf(stderr, "Error: Unable to allocate memory\n");
        fclose(file);
        exit(1);
    }

    // Read pixel data
    fread(*pixels, sizeof(unsigned char), (*width) * (*height), file);
    fclose(file);
}

// Function to save a PGM file
void savePGMub(const char *filename, unsigned char *pixels, unsigned int width, unsigned int height) {
    FILE *file = fopen(filename, "wb");
    if (!file) {
        fprintf(stderr, "Error: Unable to create file %s\n", filename);
        exit(1);
    }

    // Write PGM header
    fprintf(file, "P5\n");
    fprintf(file, "%u %u\n", width, height);
    fprintf(file, "255\n");

    // Write pixel data
    fwrite(pixels, sizeof(unsigned char), width * height, file);
    fclose(file);
}

void h_blur(unsigned char *arr, unsigned char *result, int width, int height) {
    int offset = 1 * width;
    for (int row = 1; row < height - 1; row++) {
        for (int col = 1; col < width - 1; col++) {
            int sum = 0;
            for (int j = -1; j <= 1; j++) {
                for (int i = -1; i <= 1; i++) {
                    sum += arr[(row + j) * width + (col + i)];
                }
            }
            result[offset + col] = sum / 9; // Total number of pixels in the neighborhood
        }
        offset += width;
    }
}

__global__ void d_blur(unsigned char *arr, unsigned char *result, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < 1 || col < 1 || row >= height - 1 || col >= width - 1)
        return;

    int sum = 0;
    for (int j = -1; j <= 1; j++) {
        for (int i = -1; i <= 1; i++) {
            int color = arr[(row + j) * width + (col + i)];
            sum += color;
        }
    }
    result[row * width + col] = sum / 9;
}

int main(int argc, char **argv) {
    unsigned char *d_resultPixels;
    unsigned char *h_resultPixels;
    unsigned char *h_pixels = NULL;
    unsigned char *d_pixels = NULL;
    unsigned int width, height;

    // Define array of image paths
    const char* imagePaths[] = {"mountains.pgm", "nature.pgm", "tower.pgm"};
    int numImages = sizeof(imagePaths) / sizeof(char*);

    // Loop through each image
    for (int imgIndex = 0; imgIndex < numImages; imgIndex++) {
        char srcPath[100];
        char h_ResultPath[120];
        char d_ResultPath[120];

        // Construct file paths for input and output
        sprintf(srcPath, "%s", imagePaths[imgIndex]);
        sprintf(h_ResultPath, "h_%s_meanblur.pgm", srcPath);
        sprintf(d_ResultPath, "d_%s_meanblur.pgm", srcPath);

        // Load the input image and get dimensions
        loadPGMub(srcPath, &h_pixels, &width, &height);

        int ImageSize = sizeof(unsigned char) * width * height;

        // Allocate memory for results on host and device
        h_resultPixels = (unsigned char *)malloc(ImageSize);
        hipMalloc((void **)&d_pixels, ImageSize);
        hipMalloc((void **)&d_resultPixels, ImageSize);
        hipMemcpy(d_pixels, h_pixels, ImageSize, hipMemcpyHostToDevice);

        // Timing CPU processing
        clock_t starttime = clock();
        h_blur(h_pixels, h_resultPixels, width, height);
        clock_t endtime = clock();
        double interval = (endtime - starttime) / (double)CLOCKS_PER_SEC;
        printf("CPU execution time = %f ms for %s\n", interval * 1000, srcPath);
        savePGMub(h_ResultPath, h_resultPixels, width, height);

        // Define block and grid size for CUDA
        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        // Prepare to time GPU processing
        float gpu_time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Perform blur on GPU
        d_blur<<<grid, block>>>(d_pixels, d_resultPixels, width, height);
        hipDeviceSynchronize();

        // Stop timing after synchronization
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpu_time, start, stop);
        printf("GPU execution time = %f ms for %s\n", gpu_time, srcPath);

        hipMemcpy(h_resultPixels, d_resultPixels, ImageSize, hipMemcpyDeviceToHost);
        savePGMub(d_ResultPath, h_resultPixels, width, height);

        // Clean up
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_pixels);
        hipFree(d_resultPixels);
        free(h_pixels);
        free(h_resultPixels);
    }

    // Prompt to exit
    printf("Press enter to exit ...\n");
    getchar();

    return 0;
}